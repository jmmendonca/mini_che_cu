#include "hip/hip_runtime.h"

bool use_stiff = .True.;

void mini_ch_dlsode(double T_in, double P_in, double t_end, double * VMR, char * network){

    int  ncall;
    double P_cgs;

    // Time controls
    double t_begin, t_now, t_old;

    // DLSODE variables
    double  y[n_sp], y_old[n_sp];
    double *rwork, *rtol, *atol;
    integer, allocatable, dimension(:) :: iwork
    integer :: itol, itask, istate, iopt, mf
    integer :: rworkdim, iworkdim

    !! Find the number density of the atmosphere
    P_cgs = P_in * 10.0_dp   ! Convert pascal to dyne cm-2
    nd_atm = P_cgs/(kb*T_in)  ! Find initial number density [cm-3] of atmosphere

    allocate(Keq(n_reac), re_f(n_reac), re_r(n_reac))

    ! First find the reverse reaction coefficents (Keq)
    call reverse_reactions(T_in)
    ! Find the forward, backward and net reaction rates
    call reaction_rates(T_in, P_cgs, nd_atm)

    !! Find initial number density of all species from VMR
    y(:) = nd_atm * VMR(:)

    ! -----------------------------------------
    ! ***  parameters for the DLSODE solver  ***
    ! -----------------------------------------

    itask = 1
    istate = 1
    iopt = 1

    ! Method flag
    if (use_stiff .eqv. .True.) then
      ! Problem is stiff (usual)
      ! mf = 21 - full jacobian matrix with jacobian save
      ! mf = 22 - internal calculated jacobian
      mf = 21
      rworkdim = 22 +  9*n_sp + n_sp**2
      iworkdim = 20 + n_sp
      allocate(rtol(n_sp), atol(n_sp), rwork(rworkdim), iwork(iworkdim))

      itol = 4
      rtol(:) = 1.0e-3_dp           ! Relative tolerances for each scalar
      atol(:) = 1.0e-99_dp               ! Absolute tolerance for each scalar (floor value)

      rwork(:) = 0.0_dp
      iwork(:) = 0

      rwork(1) = 0.0_dp               ! Critical T value (don't integrate past time here)
      rwork(5) = 0.0_dp              ! Initial starting timestep (start low, will adapt in DVODE)
      rwork(6) = 0.0_dp       ! Maximum timestep

      iwork(5) = 0               ! Max order required
      iwork(6) = 100000               ! Max number of internal steps
      iwork(7) = 1                ! Number of error messages

    else
      ! Problem is not too stiff (not typical)
      ! mf = 11 - full jacobian matrix with jacobian save
      mf = 11
      rworkdim = 22 + 16*n_sp + 2*n_sp**2
      iworkdim = 30 + n_sp
      allocate(rtol(n_sp), atol(n_sp), rwork(rworkdim), iwork(iworkdim))
      itol = 4
      rtol(:) = 1.0e-3_dp
      atol(:) = 1.0e-99_dp

      rwork(1) = t_end

      rwork(5:10) = 0.0_dp
      iwork(5:10) = 0

      rwork(5) = 1.0e-99_dp
      rwork(6) = t_end
      iwork(6) = 100000
    end if

    t_begin = 0.0_dp
    t_now = t_begin

    ! Set the printing flag
    ! 0 = no printing, 1 = printing
    call xsetf(1)

    ncall = 0

    do while (t_now < t_end)

      y_old(:) = y(:)
      t_old = t_now

      select case(network)
      case('HO')
        call DLSODE (RHS_update, n_sp, y, t_now, t_end, itol, rtol, atol, itask, &
        & istate, iopt, rwork, rworkdim, iwork, iworkdim, jac_HO, mf)
      case('CHO')
        call DLSODE (RHS_update, n_sp, y, t_now, t_end, itol, rtol, atol, itask, &
        & istate, iopt, rwork, rworkdim, iwork, iworkdim, jac_CHO, mf)
      case('NCHO')
        call DLSODE (RHS_update, n_sp, y, t_now, t_end, itol, rtol, atol, itask, &
        & istate, iopt, rwork, rworkdim, iwork, iworkdim, jac_NCHO, mf)
      case default
        print*, 'Invalid network provided: ', trim(network)
        stop
      end select

      ! call check_con(n_sp,y(:),y_old(:),t_now,t_old,con)
      ! if (con .eqv. .True.) then
      !   exit
      ! end if

      ncall = ncall + 1

      if (mod(ncall,50) == 0) then
        istate = 1
      else  if (istate == -1) then
        istate = 2
      else if (istate < -1) then
        print*, 'dlsode: ', istate
        exit
      end if

    end do

    VMR(:) = y(:)/nd_atm

    deallocate(Keq, re_r, re_f, rtol, atol, rwork, iwork)

  end subroutine mini_ch_dlsode

  subroutine RHS_update(NEQ, time, y, f, rpar, ipar)
    implicit none

    integer, intent(in) ::  NEQ
    real(dp), intent(inout) :: time
    real(dp), dimension(NEQ), intent(inout) :: y
    real(dp), dimension(NEQ), intent(inout) :: f
    real(dp), intent(inout) :: rpar
    integer, intent(inout) :: ipar

    integer :: i, k, j
    real(dp) :: msum, msum2, frate, rrate
    real(dp), dimension(n_reac) :: net_pr, net_re
    real(dp), dimension(NEQ) :: f_pr, f_re, t_pr, t_re
    real(dp), dimension(NEQ) :: c_pr, c_re

    ! Calculate the rate of change of number density for all species [cm-3/s] this is the f vector
    f_pr(:) = 0.0_dp
    c_pr(:) = 0.0_dp
    f_re(:) = 0.0_dp
    c_re(:) = 0.0_dp

    ! Loop through reactions add rates to the f array
    do i = 1, n_reac
      ! Do the forward and backward flux calculation for each speices in the reaction

      ! Find number density multiple for reactants in reaction
      msum = y(re(i)%gi_re(1))
      do k = 2, re(i)%n_re
         msum = msum * y(re(i)%gi_re(k))
      end do

      ! Find number density multiple for products in reaction
      msum2 = y(re(i)%gi_pr(1))
      do k = 2, re(i)%n_pr
         msum2 = msum2 * y(re(i)%gi_pr(k))
      end do

      if (re(i)%re_t == 3) then
        ! Mutliply both msum and msum2 by atmosphere nd if neutral body involved
        msum = msum * nd_atm
        msum2 = msum2 * nd_atm
      end if

      frate = msum * re_f(i)
      rrate = msum2 * re_r(i)

      net_pr(i) = frate - rrate
      net_re(i) = -net_pr(i)

      !! Perform the Kahan-Babushka-Neumaier compensation sum algorithm
      ! This is slightly slower than peicewise addition for small timesteps, but faster for larger timesteps, 
      ! and more general (should work for all networks)

      !! Add the product rates
      do j = 1, re(i)%n_pr
        t_pr(re(i)%gi_pr(j)) = f_pr(re(i)%gi_pr(j)) + net_pr(i)
        if (abs(f_pr(re(i)%gi_pr(j))) >= abs(net_pr(i))) then
          c_pr(re(i)%gi_pr(j)) = c_pr(re(i)%gi_pr(j)) + (f_pr(re(i)%gi_pr(j)) - t_pr(re(i)%gi_pr(j))) + net_pr(i)
        else
          c_pr(re(i)%gi_pr(j)) = c_pr(re(i)%gi_pr(j)) + (net_pr(i) - t_pr(re(i)%gi_pr(j))) + f_pr(re(i)%gi_pr(j))
        end if
        f_pr(re(i)%gi_pr(j)) = t_pr(re(i)%gi_pr(j))
      end do
      f_pr(re(i)%gi_pr(:)) =  f_pr(re(i)%gi_pr(:)) + c_pr(re(i)%gi_pr(:))

      !! Add the reactant rates
      do j = 1, re(i)%n_re
        t_re(re(i)%gi_re(j)) = f_re(re(i)%gi_re(j)) + net_re(i)
        if (abs(f_re(re(i)%gi_re(j))) >= abs(net_re(i))) then
          c_re(re(i)%gi_re(j)) = c_re(re(i)%gi_re(j)) + (f_re(re(i)%gi_re(j)) - t_re(re(i)%gi_re(j))) + net_re(i)
        else
          c_re(re(i)%gi_re(j)) = c_re(re(i)%gi_re(j)) + (net_re(i) - t_re(re(i)%gi_re(j))) + f_re(re(i)%gi_re(j))
        end if
        f_re(re(i)%gi_re(j)) = t_re(re(i)%gi_re(j))
      end do
      f_re(re(i)%gi_re(:)) =  f_re(re(i)%gi_re(:)) + c_re(re(i)%gi_re(:))

    end do

    !! Sum product and reactant rates to get net rate for species
    f(:) = f_pr(:) + f_re(:)
 
  end subroutine RHS_update

  subroutine jac_dummy (NEQ, X, Y, ML, MU, PD, NROWPD)
    integer, intent(in) :: NEQ, ML, MU, NROWPD
    real(dp), intent(in) :: X
    real(dp), dimension(NEQ), intent(in) :: Y
    real(dp), dimension(NROWPD, NEQ), intent(inout) :: PD
  end subroutine jac_dummy

void jac_NCHO(N, X, Y, ML, MU, DFY, NROWPD){

    dfy[0][0] = -f[0]*y[1] - f[1]*y[4] - r[2]*y[3];
    dfy[0][1] = -f[0]*y[0] + f[2]*y[6];
    dfy[0][2] = r[0]*y[3];
    dfy[0][3] = r[0]*y[2] + re_r[1]*y[5] - re_r[2]*y[0];
    dfy[0][4] = -f[1]*y[0];
    dfy[0][5] = r[1]*y[3];
    dfy[0][6] = f[2]*y[1];
    dfy[0][7] = 0.0;
    dfy[0][8] = 0.0;
    dfy[0][9] = 0.0;
    dfy[0][10] = 0.0;
    dfy[0][11] = 0.0;

    dfy[1][0] = -f[0]*y[1] + r[2]*y[3];
    dfy[1][1] = -nd_atm*r[3] - 9.0*r[4]*y[1]*y[1]*y[4] - 
            9.0*r[5]*y[1]*y[1]*y[8] - 9.0*r[7]*y[10]*y[1]*y[1] - 
            9.0*r[8]*y[11]*y[1]*y[1] - f[0]*y[0] - f[2]*y[6];
    dfy[1][2] = r[0]*y[3] + 3.0*f[4]*y[7];
    dfy[1][3] = 2.0*nd_atm*f[3]*y[3] + r[0]*y[2] + r[2]*y[0];
    dfy[1][4] = -3.0*r[4]*y[2]*y[2]*y[2];
    dfy[1][5] = 0.0;
    dfy[1][6] = -f[2]*y[1];

    dfy[1][7] = 6.0*f[5]*y[7] + 3.0*f[8]*y[9] + 3.0*f[4]*y[2];
    dfy[1][8] = -3.0*r[5]*y[1]*y[1]*y[1];
    dfy[1][9] = 6.0*f[7]*y[9] + 3.0*f[8]*y[7];
    dfy[1][10] = -3.0*r[7]*y[1]*y[1]*y[1];
    dfy[1][11] = -3.0*r[8]*y[1]*y[1]*y[1];

    dfy[2][0] = f[0]*y[1];
    dfy[2][1] = 3.0*r[4]*y[1]*y[1]*y[4] + f[0]*y[0];
    dfy[2][2] = -r[6]*y[8] - re_r[9]*y[11] - r[0]*y[3] - f[4]*y[7];
    dfy[2][3] = -r[0]*y[2];
    dfy[2][4] = r[4]*y[1]*y[1]*y[1] + f[6]*y[7] + f[9]*y[9];
    dfy[2][5] = 0.0;
    dfy[2][6] = 0.0;
    dfy[2][7] = f[6]*y[4] - f[4]*y[2];
    dfy[2][8] = -re_r[6]*y[2];
    dfy[2][9] = f[9]*y[4];
    dfy[2][10] = 0.0;
    dfy[2][11] = -r[9]*y[2];

    dfy[3][0] = f[0]*y[1] + f[1]*y[4] - r[2]*y[3];
    dfy[3][1] = 2.0*nd_atm*r[3] + f[0]*y[0] + f[2]*y[6];
    dfy[3][2] = -r[0]*y[3];
    dfy[3][3] = -4.0*nd_atm*f[3]*y[3] - re_r[0]*y[2] - 
            r[1]*y[5] - r[2]*y[0];
    dfy[3][4] = f[1]*y[0];
    dfy[3][5] = -r[1]*y[3];
    dfy[3][6] = f[2]*y[1];
    dfy[3][7] = 0.0;
    dfy[3][8] = 0.0;
    dfy[3][9] = 0.0;
    dfy[3][10] = 0.0;
    dfy[3][11] = 0.0;

    dfy[4][0] = -f[1]*y[4];
    dfy[4][1] = -3.0*r[4]*y[1]*y[1]*y[4];
    dfy[4][2] = r[6]*y[8] + r[9]*y[11] + f[4]*y[7];
    dfy[4][3] = r[1]*y[5];
    dfy[4][4] = -r[4]*y[1]*y[1]*y[1] - f[6]*y[7] - f[9]*y[9] - f[1]*y[0];
    dfy[4][5] = r[1]*y[3];
    dfy[4][6] = 0.0;
    dfy[4][7] = -f[6]*y[4] + f[4]*y[2];
    dfy[4][8] = r[6]*y[2];
    dfy[4][9] = -f[9]*y[4];
    dfy[4][10] = 0.0;
    dfy[4][11] = r[9]*y[2];

    dfy[5][0] = f[1]*y[4];
    dfy[5][1] = 0.0;
    dfy[5][2] = 0.0;
    dfy[5][3] = -r[1]*y[5];
    dfy[5][4] = f[1]*y[0];
    dfy[5][5] = -r[1]*y[3];
    dfy[5][6] = 0.0;
    dfy[5][7] = 0.0;
    dfy[5][8] = 0.0;
    dfy[5][9] = 0.0;
    dfy[5][10] = 0.0;
    dfy[5][11] = 0.0;

dfy[6][0] = r[2]*y[5];
dfy[6][1] = -r[2]*y[6];
dfy[6][2] = 0.0;
dfy[6][3] = r[2]*y[0];
dfy[6][4] = 0.0;
dfy[6][5] = 0.0;
dfy[6][6] = -f[2]*y[1];
dfy[6][7] = 0.0;
dfy[6][8] = 0.0;
dfy[6][9] = 0.0;
dfy[6][10] = 0.0;
dfy[6][11] = 0.0;

dfy[7][0] = 0.0;
dfy[7][1] = 3.0*r[4]*y[1]*y[1]*y[4] + 6.0*r[5]*y[1]*y[2]*y[8] + 
            3.0*r[8]*y[1]*y[1]*y[11];
dfy[7][2] = r[6]*y[8] - f[4]*y[7];
dfy[7][3] = 0.0;
dfy[7][4] = r[4]*y[1]*y[1]*y[1] - f[6]*y[7];
dfy[7][5] = 0.0;
dfy[7][6] = 0.0;
dfy[7][7] = -4.0*f[5]*y[7] - f[6]*y[4] - f[8]*y[9] - f[4]*y[2];
dfy[7][8] = 2.0*r[5]*y[1]*y[1]*y[1] + r[6]*y[2];
dfy[7][9] = -f[8]*y[7];
dfy[7][10] = 0.0;
dfy[7][11] = r[8]*y[1]*y[1]*y[1];

dfy[8][0] = 0.0;
dfy[8][1] = -3.0*r[5]*y[1]*y[1]*y[8];
dfy[8][2] = -r[6]*y[8];
dfy[8][3] = 0.0;
dfy[8][4] = f[6]*y[7];
dfy[8][5] = 0.0;
dfy[8][6] = 0.0;
dfy[8][7] = 2.0*f[5]*y[7] + f[6]*y[4];
dfy[8][8] = -r[5]*y[1]*y[1] - r[6]*y[2];
dfy[8][9] = 0.0;
dfy[8][10] = 0.0;
dfy[8][11] = 0.0;

dfy[9][0] = 0.0;
dfy[9][1] = 6.0*r[7]*y[10]*y[1]*y[1] + 3.0*r[8]*y[11]*y[1]*y[1];
dfy[9][2] = r[9]*y[11];
dfy[9][3] = 0.0;
dfy[9][4] = -f[9]*y[9];
dfy[9][5] = 0.0;
dfy[9][6] = 0.0;
dfy[9][7] = -f[9]*y[9];
dfy[9][8] = 0.0;
dfy[9][9] = -4.0*f[7]*y[9] - r[8]*y[7] - f[9]*y[4];
dfy[9][10] = 2.0*r[7]*y[1]*y[1]*y[1];
dfy[9][12] = r[8]*y[1]*y[1]*y[1] + r[9]*y[2];

dfy[10][0] = 0.0;
dfy[10][1] = -3.0*r[7]*y[1]*y[1]*y[10];
dfy[10][2] = 0.0;
dfy[10][3] = 0.0;
dfy[10][4] = 0.0;
dfy[10][5] = 0.0;
dfy[10][6] = 0.0;
dfy[10][7] = 0.0;
dfy[10][8] = 0.0;
dfy[10][9] = 2.0*f[7]*y[9];
dfy[10][10] = -r[7]*y[1]*y[1]*y[1];
dfy[10][11] = 0.0;

dfy[11][0] = 0.0;
dfy[11][1] = -3.0*r[8]*y[11]*y[1]*y[1];
dfy[11][2] = -r[9]*y[11];
dfy[11][3] = 0.0;
dfy[11][4] = f[9]*y[9];
dfy[11][5] = 0.0;
dfy[11][6] = 0.0;
dfy[11][7] = f[8]*y[9];
dfy[11][8] = 0.0;
dfy[11][9] = f[8]*y[7] + f[9]*y[4];
dfy[11][10] = 0.0;
dfy[11][11] = -r[8]*y[1]*y[1]*y[1] - r[9]*y[2];
}

void jac_CHO(N, X, Y, ML, MU, DFY, NROWPD){

    dfy[0][0] = -f[0]*y[1] - f[1]*y[4] - r[2]*y[3]
    dfy[0][1] = -f[0]*y[0] + f[2]*y[6]
    dfy[0][2] = r[0]*y[3]
    dfy[0][3] = r[0]*y[2] + r[1]*y[5] - r[2]*y[0]
    dfy[0][4] = -f[1]*y[0]
    dfy[0][5] = r[1]*y[3]
    dfy[0][6] = f[2]*y[1]
    dfy[0][7] = 0.0
    dfy[0][8] = 0.0
    
    dfy[1][0] = -f[0]*y[1] + r[2]*y[3]
    dfy[1][1] = -nd_atm*r[3] - 9.0*r[4]*y[1]*y[1]*y[4] - 
      9.0*r[5]*y[1]*y[1]*y[8] - f[0]*y[0] - f[2]*y[6]
    dfy[1][2] = r[0]*y[3] + 3.0*f[4]*y[7]
    dfy[1][3] = 2.0*nd_atm*f[3]*y[3] + r[0]*y[2] + r[2]*y[0]
    dfy[1][4] = -3.0*r[4]*y[1]*y[1]*y[1]
    dfy[1][5] = 0.0
    dfy[1][6] = -f[2]*y[1]
    dfy[1][7] = 6.0*f[5]*y[7] + 3.0*f[4]*y[2]
    dfy[1][8] = -3.0*r[5]*y[1]*y[1]*y[1]
    
    dfy[2][0] = f[0]*y[1]
    dfy[2][1] = 3.0*r[4]*y[1]*y[1]*y[4] + f[0]*y[0]
    dfy[2][2] = -r[6]*y[8] - r[0]*y[3] - f[4]*y[7]
    dfy[2][3] = -r[0]*y[2]
    dfy[2][4] = r[4]*y[1]*y[1]*y[1] + f[6]*y[7]
    dfy[2][5] = 0.0
    dfy[2][6] = 0.0
    dfy[2][7] = f[6]*y[4] - f[4]*y[2]
    dfy[2][8] = -r[6]*y[2]
    
    dfy[3][0] = f[0]*y[1] + f[1]*y[6] - r[2]*y[3]
    dfy[3][1] = 2.0*nd_atm*r[3] + f[0]*y[0] + f[2]*y[6]
    dfy[3][2] = -r[0]*y[3]
    dfy[3][3] = -4.0*nd_atm*f[3]*y[3] - r[0]*y[2] - r[1]*y[5] - r[2]*y[0]
    dfy[3][4] = f[1]*y[0]
    dfy[3][5] = -r[1]*y[3]
    dfy[3][6] = f[2]*y[1]
    dfy[3][7] = 0.0
    dfy[3][8] = 0.0
   
    dfy[4][0] = -f[1]*y[4]
    dfy[4][1] = -3.0*r[4]*y[1]*y[1]*y[4]
    dfy[4][2] = r[6]*y[8] + f[4]*y[7];
    dfy[4][3] = r[1]*y[5];
    dfy[4][4] = -3.0*r[4]*y[2]*y[2]*y[5] - f[6]*y[7] - f[1]*y[6];
    dfy[4][5] = r[2]*y[5];
    dfy[4][6] = 0.0;
    dfy[4][7] = -f[6]*y[4] + f[4]*y[2];
    dfy[4][8] = r[6]*y[2];
   
    dfy[5][0] = f[1]*y[4];
    dfy[5][1] = 0.0;
    dfy[5][2] = 0.0;
    dfy[5][3] = -r[1]*y[5];
    dfy[5][4] = f[1]*y[0];
    dfy[5][5] = -r[1]*y[3];
    dfy[5][6] = 0.0;
    dfy[5][7] = 0.0;
    dfy[5][8] = 0.0;

    dfy[5][0] = r[2]*y[3];  
    dfy[5][1] = 0.0;
    dfy[5][2] = 0.0;
    dfy[5][3] = -r[1]*y[5];
    dfy[5][4] = f[1]*y[0];
    dfy[5][5] = -r[1]*y[3];
    dfy[5][6] = 0.0;
    dfy[5][7] = 0.0;
    dfy[5][8] = 0.0;
  
    dfy[6][0] = r[2]*y[3];
    dfy[6][1] = -f[2]*y[6];
    dfy[6][2] = 0.0;
    dfy[6][3] = r[2]*y[0];
    dfy[6][4] = 0.0;
    dfy[6][5] = 0.0;
    dfy[6][6] = -f[2]*y[1];
    dfy[6][7] = 0.0;
    dfy[6][8] = 0.0;
  
    dfy[7][0] = 0.0;
    dfy[7][1] = 3.0*r[4]*y[1]*y[1]*y[4] + 6.0*r[5]*y[2]*y[1]*y[8];
    dfy[7][2] = r[6]*y[8] - f[4]*y[7];
    dfy[7][3] = 0.0;
    dfy[7][4] = r[4]*y[1]*y[1]*y[1] - f[6]*y[7];
    dfy[7][5] = 0.0;
    dfy[7][6] = 0.0;
    dfy[7][7] = -4.0*f[5]*y[7] - f[6]*y[4] - f[4]*y[2];
    dfy[7][8] = 2.0*r[5]*y[1]*y[1]*y[1] + r[6]*y[2];
  
    dfy[8][0] = 0.0;
    dfy[8][1] = -3.0*r[5]*y[1]*y[1]*y[8];
    dfy[8][2] = -r[6]*y[8];
    dfy[8][3] = 0.0;
    dfy[8][4] = f[6]*y[7];
    dfy[8][5] = 0.0;
    dfy[8][6] = 0.0;
    dfy[8][7] = 2.0*f[5]*y[7] + f[6]*y[4];
    dfy[8][8] = -r[5]*y[1]*y[1]*y[1] - r[6]*y[2];
}

void jac_HO(double *N, double *X, Y, ML, MU, DFY, NROWPD)

    dfy[0][0] = -f[0]*y[1] - r[1]*y[3];
    dfy[0][1] = -f[0]*y[0] + f[1]*y[4];
    dfy[0][2] = r[0]*y[3];
    dfy[0][3] = r[0]*y[2] - r[1]*y[0];
    dfy[0][4] = f[1]*y[1];
   
    dfy[1][0] = -f[0]*y[1] + r[1]*y[3];
    dfy[1][1] = -nd_atm*r[2] - f[0]*y[0] - f[1]*y[4];
    dfy[1][2] = r[0]*y[3];
    dfy[1][3] = 2.0*nd_atm*f[2]*y[3] + r[0]*y[2] + r[1]*y[0];
    dfy[1][4] = -f[1]*y[1];
  
    dfy[2][0] = f[0]*y[1];
    dfy[2][1] = f[0]*y[0];
    dfy[2][2] = -r[0]*y[3];
    dfy[2][3] = -r[0]*y[2];
    dfy[2][4] = 0.0;

    dfy[3][0] = f[0]*y[1] - r[1]*y[3];
    dfy[3][1] = 2.0*nd_atm*r[2] + f[0]*y[0] + f[1]*y[4];
    dfy[3][2] = -r[0]*y[3];
    dfy[3][3] = -4.0*nd_atm*f[2]*y[3] - r[0]*y[2] - r[1]*y[0];
    dfy[3][4] = f[1]*y[1];

    dfy[4][0] = r[1]*y[3];
    dfy[4][1] = -f[1]*y[4];
    dfy[4][2] = 0.0;
    dfy[4][3] = r[1]*y[0];
    dfy[4][4] = -f[1]*y[1];
}
