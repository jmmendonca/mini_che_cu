
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <numeric>
#include <cstring>

struct MiniChemParams {
    double T_in;
    double P_in;
    double t_step;
    int n_step;
    int n_sp;
    char data_file[200];
    char sp_file[200];
    char network[200];
    char net_dir[200];
    char met[200];
};

struct MiniChemVMRParams {
    int CE_IC;
    char IC_file[200];
    double* VMR_IC;
};

void miniChDlsode(double T_in, double P_in, double t_step, double* VMR, char* network) {
    // Implementation of the mini_ch_dlsode function
    // ... (you need to implement this function)
}










int main() {
    std::ifstream file("mini_chem.nml");
    MiniChemParams params;
    MiniChemVMRParams VMRParams;

    //params.network = "OH";
    //params.network = "CHO";
    std::strcpy(params.network, "NCHO");

    params.T_in = 1500.0;
    params.P_in = 1.0e6;

    params.t_step = 60;
    params.n_step = 500000;

    params.n_sp = 12;

    std::strcpy(params.data_file, "chem_data/mini_chem_data_NCHO.txt");
    std::strcpy(params.sp_file, "chem_data/mini_chem_sp_NCHO.txt");
    std::strcpy(params.net_dir, "chem_data/1x/");
    std::strcpy(params.met, "1x");

    VMRParams.CE_IC = 1;
    std::strcpy(VMRParams.IC_file, "chem_data/IC/mini_chem_IC_FastChem_1x.txt");

    //VMRParams.VMR_IC = {0.0, 0.8, 0.0, 0.0, 0.2};
    //VMRParams.VMR_IC = {0.0, 0.8, 0.0, 0.0, 0.1, 0.0, 0.1, 0.0, 0.0};
    
    // Allocate memory for the array
    VMRParams.VMR_IC = new double[params.n_sp];

    // Initialize the array with values
    double initialVMR[] = {0.0, 0.9975, 0.001074, 0.0, 0.0, 0.0, 0.0, 0.00059024, 0.0, 0.00014159, 0.0, 0.0};

    // Copy the values into the array
    std::copy(std::begin(initialVMR), std::end(initialVMR), VMRParams.VMR_IC);

    std::cout << "T [K], P [bar], t_step, n_step, n_sp :" << std::endl;
    std::cout << params.T_in << " " << params.P_in << " " << params.t_step << " " << params.n_step << " " << params.n_sp << std::endl;

    // Initial time
    double t_now = 0.0;

    // Read the reaction and species list
    // (you need to implement the read_react_list function)

    // Save the initial conditions to file
    // Rescale IC to 1
    double sum_VMR_IC = std::accumulate(VMRParams.VMR_IC, VMRParams.VMR_IC + params.n_sp, 0.0);
    for (int i = 0; i < params.n_sp; ++i)
        VMRParams.VMR_IC[i] /= sum_VMR_IC;

    std::cout << "IC: ";
    for (int i = 0; i < params.n_sp; ++i)
        std::cout << VMRParams.VMR_IC[i] << " ";

    std::cout << sum_VMR_IC << std::endl;

    // Give initial conditions to VMR array
    double* VMR = new double[params.n_sp];
    std::memcpy(VMR, VMRParams.VMR_IC, params.n_sp * sizeof(double));

    // Do time marching loop
    // - this loop emulates what a call to the model is like in the GCM
    for (int n = 1; n <= params.n_step; ++n) {
        // Update time
        t_now += params.t_step;

        // Time now
        std::cout << n << " " << params.n_step << " " << t_now << std::endl;

        // Scale VMR to 1
        double sum_VMR = std::accumulate(VMR, VMR + params.n_sp, 0.0);
        for (int i = 0; i < params.n_sp; ++i)
            VMR[i] /= sum_VMR;

        // Call dlsode - bdf method
        miniChDlsode(params.T_in, params.P_in, params.t_step, VMR, params.network);
        std::cout << "dlsode: ";
        for (int i = 0; i < params.n_sp; ++i)
            std::cout << VMR[i] << " ";

        std::cout << sum_VMR << std::endl;

        // Scale VMR to 1
        for (int i = 0; i < params.n_sp; ++i)
            VMR[i] /= sum_VMR;
    }

    // Clean up
    delete[] VMR;
    delete[] VMRParams.VMR_IC;

    return 0;
}
